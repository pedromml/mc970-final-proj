
#include <hip/hip_runtime.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define MASK_WIDTH 15

#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255

void check_cuda(hipError_t error, const char *filename, const int line)
{
  if (error != hipSuccess) {
    fprintf(stderr, "Error: %s:%d: %s: %s\n", filename, line,
                 hipGetErrorName(error), hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

#define CUDACHECK(cmd) check_cuda(cmd, __FILE__, __LINE__)

typedef struct {
  unsigned char red, green, blue;
} PPMPixel;

typedef struct {
  int x, y;
  PPMPixel *data;
} PPMImage;

static PPMImage *readPPM(const char *filename) {
  char buff[16];
  PPMImage *img;
  FILE *fp;
  int c, rgb_comp_color;
  fp = fopen(filename, "rb");
  if (!fp) {
    fprintf(stderr, "Unable to open file '%s'\n", filename);
    exit(1);
  }

  if (!fgets(buff, sizeof(buff), fp)) {
    perror(filename);
    exit(1);
  }

  if (buff[0] != 'P' || buff[1] != '3') {
    fprintf(stderr, "Invalid image format (must be 'P3')\n");
    exit(1);
  }

  img = (PPMImage *)malloc(sizeof(PPMImage));
  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  c = getc(fp);
  while (c == '#') {
    while (getc(fp) != '\n')
      ;
    c = getc(fp);
  }

  ungetc(c, fp);
  if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
    fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
    exit(1);
  }

  if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
    fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
    exit(1);
  }

  // if (rgb_comp_color != RGB_COMPONENT_COLOR) {
  //   fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
  //   exit(1);
  // }

  while (fgetc(fp) != '\n')
    ;
  img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

  if (!img) {
    fprintf(stderr, "Unable to allocate memory\n");
    exit(1);
  }

  int r, g, b;
  int index = 0;
  while(fscanf(fp, "%d %d %d", &b, &g, &r) != EOF){
    PPMPixel p = {b, g, r};
    img->data[index] = p;
    index +=1;

  }

  // if (fread(img->data, sizeof(PPMPixel) * img->x / 2, img->y, fp) != img->y) {
  //   fprintf(stderr, "Error loading image '%s'\n", filename);
  //   exit(1);
  // }

  fclose(fp);
  return img;
}
void writePPM(PPMImage *img) {
  fprintf(stdout, "P3\n");
  // fprintf(stdout, "# %s\n", COMMENT);
  fprintf(stdout, "%d %d\n", img->x, img->y);
  fprintf(stdout, "%d\n", 1);
  // printf(sizeof(img->data) / sizeof(img->data[0]));
  // printf(sizeof(img->data));


  for(int index = 0; index < img->x * img->y; index++){

    fprintf(stdout, "%d %d %d  ", img->data[index].blue, img->data[index].green, img->data[index].red);
    if(((index + 1) % img->x == 0)){
      fprintf(stdout, "\n");
    }
  }
  // fwrite(img->data, sizeof(PPMPixel) * img->x, img->y, stdout);
  fclose(stdout);
}

// Implement this!
__global__ void smoothing_kernel(PPMImage* image, PPMPixel* data, PPMPixel* data_copy) {
  int iteration = blockDim.x * blockIdx.x + threadIdx.x;
  int n = (image->y * image->x);
  if(iteration < n){
    int i, j, y, x;
    int live_cells_around;

    i = iteration / image->x;
    j = iteration - (i * image->x);

    for (y = i - 1; y <= (i + 1);
        y++) {
      for (x = j - 1; x <= (j + 1);
          x++) {
        if (x >= 0 && y >= 0 && y < image->y && x < image->x) {
          live_cells_around += data_copy[(y * image->x) + x].red;
        } // if
      }   // for z
    }     // for y
    if(data[(i * image->x) + j].red == 1 && live_cells_around < 2){
      data[(i * image->x) + j].red = 0;
      data[(i * image->x) + j].blue = 0;
      data[(i * image->x) + j].green = 0;
    }
    if(data[(i * image->x) + j].red == 1 && (live_cells_around >= 2 && live_cells_around <= 3)){
      data[(i * image->x) + j].red = 1;
      data[(i * image->x) + j].blue = 1;
      data[(i * image->x) + j].green = 1;
    }
    if(data[(i * image->x) + j].red == 0 && (live_cells_around == 3)){
      data[(i * image->x) + j].red = 1;
      data[(i * image->x) + j].blue = 1;
      data[(i * image->x) + j].green = 1;
    }
    if(data[(i * image->x) + j].red == 1 && (live_cells_around > 3)){
      data[(i * image->x) + j].red = 0;
      data[(i * image->x) + j].blue = 0;
      data[(i * image->x) + j].green = 0;
    }

  }
}

void Smoothing(PPMImage *image, PPMImage *image_copy) {
  // printf("Size 1: %d", sizeof(image->data) / sizeof(PPMPixel));
  // printf("Item 1: %d", image->data[0].red);
  // printf("image size: %d", image->x * image->y);
  PPMImage *image_d;
  PPMPixel *data_d;
  PPMPixel *data_copy_d;

  int data_size = image->x * image->y;

  float ms;
  hipEvent_t start, stop;
  
  CUDACHECK(hipEventCreate(&start));
  CUDACHECK(hipEventCreate(&stop));

  hipMalloc((void **) &image_d, sizeof(image));
  hipMemcpy(image_d, image, sizeof(image), hipMemcpyHostToDevice);

  hipMalloc((void **) &data_d, data_size * sizeof(PPMPixel));
  hipMemcpy(data_d, image->data, data_size * sizeof(PPMPixel), hipMemcpyHostToDevice);

  hipMalloc((void **) &data_copy_d, data_size * sizeof(PPMPixel));
  hipMemcpy(data_copy_d, image_copy->data, data_size * sizeof(PPMPixel), hipMemcpyHostToDevice);
  

  // Launch kernel and compute kernel runtime.
  // Warning: make sure only the kernel is being profiled, memcpies should be
  // out of this region.
  
  int cudaBlockSize = 512;
  int cudaBlocks = (image->x * image->y) / cudaBlockSize + 1;

  CUDACHECK(hipEventRecord(start));
  smoothing_kernel<<<cudaBlocks, cudaBlockSize>>>(image_d, data_d, data_copy_d);
  hipDeviceSynchronize();
  CUDACHECK(hipEventRecord(stop));
  CUDACHECK(hipEventSynchronize(stop));
  CUDACHECK(hipEventElapsedTime(&ms, start, stop));

  hipMemcpy(image->data, data_d, data_size * sizeof(PPMPixel), hipMemcpyDeviceToHost);
  // printf("Size 2: %d", sizeof(image->data) / sizeof(PPMPixel));
  // printf("Item 2: %d", image->data[0].red);

  hipFree(image_d);
  hipFree(data_d);
  hipFree(data_copy_d);

  // Destroy events
  CUDACHECK(hipEventDestroy(start));
  CUDACHECK(hipEventDestroy(stop));
}

int main(int argc, char *argv[]) {
  FILE *input;
  char filename[255];
  double t;

  if (argc < 2) {
    fprintf(stderr, "Error: missing path to input file\n");
    return 1;
  }

  if ((input = fopen(argv[1], "r")) == NULL) {
    fprintf(stderr, "Error: could not open input file!\n");
    return 1;
  }

  // Read input filename
  fscanf(input, "%s\n", filename);

  // Read input file
  PPMImage *image = readPPM(filename);
  PPMImage *image_output = readPPM(filename);

  // Call Smoothing Kernel
  t = omp_get_wtime();
  Smoothing(image_output, image);
  t = omp_get_wtime() - t;

  // Write result to stdout
  writePPM(image_output);

  // Print time to stderr
  fprintf(stderr, "%lf\n", t);

  // Cleanup
  free(image);
  free(image_output);

  return 0;
}
